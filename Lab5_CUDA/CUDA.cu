#include "hip/hip_runtime.h"
#include <stdio.h>
#include<ctime>
#include<iostream>
#include"hip/hip_runtime.h"

using namespace std;

int n=1024;//数据规模
float* data = NULL;//矩阵数据

//初始化矩阵数据
void init() {
	
	srand((int)time(0));
	for (int i = 0; i < n*n; i++) {
		data[i] = rand() % 100;
	}
}

//打印矩阵
void printMatrix() {
	for (int i = 0; i < n*n; i++) {
        if(i%n==0)
           cout<<endl;
        if(data[i]==-0)
            cout<<0<<"  ";
        else
            cout<<data[i]<<"  ";
	}
}


__global__ void division_kernel(float* data, int k, int N){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;//计算线程索引
    if(tid<N){
        int element = data[k*N+k];
        int temp = data[k*N+tid];
        //请同学们思考，如果分配的总线程数小于 N 应该怎么办？

        data[k*N+tid] = (float)temp/element;
    }
    
    return;
}

__global__ void eliminate_kernel(float* data, int k, int N){
    int tx = blockDim.x * blockIdx.x + threadIdx.x;
    if(tx==0)
    data[k*N+k]=1.0;//对角线元素设为 1

    int row = k+1+blockIdx.x;//每个块负责一行

    while(row<N){
        int tid = threadIdx.x;
        while(k+1+tid < N){
            int col = k+1+tid;
            float temp_1 = data[(row*N) + col];
            float temp_2 = data[(row*N)+k];
            float temp_3 = data[k*N+col];
            data[(row*N) + col] = temp_1 - temp_2*temp_3;
            tid = tid + blockDim.x;
        }
        __syncthreads();//块内同步
        if (threadIdx.x == 0){
            data[row * N + k] = 0;
        }
        row += gridDim.x;
    }
    return;
}



int main(){
    hipMallocManaged(&data,n*n*sizeof(float));//为data分配内存
    init();
   
    
    
    hipError_t ret;
    
    hipEvent_t start, stop;//计时器
    float elapsedTime = 0.0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);//开始计时
    
    for(int k=0;k<n;k++){
        size_t threads_per_block=256;
       size_t number_of_blocks=(n+threads_per_block-1)/threads_per_block;
       
        division_kernel<<<number_of_blocks,threads_per_block>>>(data,k,n);//负责除法任务的核函数
        hipDeviceSynchronize();//CPU 与 GPU 之间的同步函数
        ret = hipGetLastError();
        if(ret!=hipSuccess){
            printf("division_kernel failed, %s\n",hipGetErrorString(ret));
        }
        
       
       
       eliminate_kernel<<<number_of_blocks,threads_per_block>>>(data,k,n);//负责消去任务的核函数
       hipDeviceSynchronize();
       ret = hipGetLastError();
        if(ret!=hipSuccess){
            printf("eliminate_kernel failed, %s\n",hipGetErrorString(ret));
        }  
        
    }
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);//停止计时
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    
    
    cout<<endl;
    printf("GPU_LU:%f ms\n", elapsedTime);
    
    hipFree ( data ) ; //释 放 data 内 存

    return 0;
}
